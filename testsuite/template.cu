////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* Template project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>


////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);


////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
////////////////////////////////////////////////////////////////////////////////
__global__ void testKernel()
{
    // shared memory
    // the size is determined by the host application
    extern  __shared__  float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    // access number of threads in this block
    const unsigned int num_threads = blockDim.x;

}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv)
{
    bool bTestResult = true;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    //int devID = findCudaDevice(argc, (const char **)argv);

    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // execute the kernel
    testKernel<<< num_threads, mem_size >>>();

    return;
}
